#include "hip/hip_runtime.h"
//Topic Recognition P5 

//include Header and Libraries files required
//note: boost library must be installed and included in the Additional Include Directories of the project Properties for this code to work.


#include<iostream>
#include<math.h>
#include<conio.h>
#include<stdlib.h>
#include<vector>
#include<algorithm>
#include<map>
#include<iterator>
#include <fstream>
#include <streambuf>
#include<string>
#include <dirent.h>
#include <boost/algorithm/string.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <random>


//setting the parameters for size of HD vectors and N-Grams
int D = 10000;
int N = 3;

//generate random HyperVectors
std::vector<int> genRandomHV()
{
	std::vector<int> randomIndex(D);
	std::vector<int> randomHV(D);
	std::mt19937 r{ std::random_device{}() };
	if ((D % 2) == 1)
	{
		std::cout << "Dimension is odd";
	}
	else
	{
		for (int i = 0; i < D; i++)
			randomIndex[i] = i;
		std::shuffle(randomIndex.begin(), randomIndex.end(), r);
		for (int i = 0;i < D / 2;i++)
			randomHV[randomIndex[i]] = 1;
		for (int i = D / 2;i < D;i++)
			randomHV[randomIndex[i]] = -1;
	}
	
	return randomHV;
}

//create the Item Memory from which the values will be derived
std::map<char, std::vector<int>> createItemMemory(std::map<char, std::vector<int>> iM)
{
	
		
	iM['a'] = genRandomHV();
	iM['b'] = genRandomHV();
	iM['c'] = genRandomHV();
	iM['d'] = genRandomHV();
	iM['e'] = genRandomHV();
	iM['f'] = genRandomHV();
	iM['g'] = genRandomHV();
	iM['h'] = genRandomHV();
	iM['i'] = genRandomHV();
	iM['j'] = genRandomHV();
	iM['k'] = genRandomHV();
	iM['l'] = genRandomHV();
	iM['m'] = genRandomHV();
	iM['n'] = genRandomHV();
	iM['o'] = genRandomHV();
	iM['p'] = genRandomHV();
	iM['q'] = genRandomHV();
	iM['r'] = genRandomHV();
	iM['s'] = genRandomHV();
	iM['t'] = genRandomHV();
	iM['u'] = genRandomHV();
	iM['v'] = genRandomHV();
	iM['w'] = genRandomHV();
	iM['x'] = genRandomHV();
	iM['y'] = genRandomHV();
	iM['z'] = genRandomHV();
	iM[char(32)] = genRandomHV();
	
	return iM;
}

//Retrieve the Character's corresponding Hyper vector
std::vector<int> lookUpitemMemory(std::map<char, std::vector<int>> iM, char key)
{
	std::vector<int> randomHV(D);
	randomHV = iM[key];
	return randomHV;
}

//Finding similarity between Two Hyper vectors
double jaccard_similarity(std::vector<int> A, std::vector<int> B) {
	double m[4] = { 0,0,0,0 };
	int check;
	for (unsigned int i = 0; i < D; i++) {
		check = A[i] + B[i];
		if (check == 2) {
			m[3]++;
		}
		else if (check == 0) {
			if (B[i] == 1) {
				m[1]++;
			}
			else {
				m[2]++;
			}
		}
		else if (check == -2) {
			m[0]++;
		}
	}
	return (m[3] / (m[1] + m[2] + m[3]));
}

//Converting the vectors into values +1 and -1 before comparison(jaccard)
std::vector<int> binarizeHV(std::vector<int> langHV)
{
	int threshold = 0;

	for (int i = 0; i < D; i++)
	{
		if (langHV[i] > threshold)
		{
			langHV[i] = 1;
		}
		else
		{
			langHV[i] = -1;
		}
	}
	
	return langHV;
}

//Main part of the program: Generates the hyper vector for each input text file
//thrust algorithm is used which allocates the memory in the device (i.e, GPU) and performs all operations in the GPU
std::vector<int> computeSumHV(std::map<char, std::vector<int>> iM, size_t bufferSize, std::string  buffer)
{
	thrust::device_vector<int> st_block2(D, 1);
	thrust::device_vector<int> st_block3(D, 1);
	thrust::device_vector<int> st_block4(D, 1);
	thrust::device_vector<int> st_block5(D, 1);
	thrust::device_vector<int> st_block6(D, 1);
	thrust::device_vector<int> st_block7(D, 1);
	thrust::device_vector<int> block0(D, 1);
	thrust::device_vector<int> block1(D, 1);
	thrust::device_vector<int> block2(D, 1);
	thrust::device_vector<int> block3(D, 1);
	thrust::device_vector<int> block4(D, 1);
	thrust::device_vector<int> block5(D, 1);
	thrust::device_vector<int> block6(D, 1);
	thrust::device_vector<int> block7(D, 1);
	thrust::device_vector<int> nGrams(D, 1);
	thrust::device_vector <int > d_sumHV(D, 0);
	std::vector<int> sumHV(D, 0);
	if (N == 3)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];
			
			std::cout << key;

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());

			st_block2 = lookUpitemMemory(iM, key);

			thrust::copy(st_block2.begin(), st_block2.end(), block2.begin());

			if (j >= 2)
			{
				thrust::copy(block2.begin(), block2.end(), nGrams.begin());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());

				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());

			}

		}

	}

	else if (N == 4)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());

			st_block3 = lookUpitemMemory(iM, key);

			thrust::copy(st_block3.begin(), st_block3.end(), block3.begin());

			if (j >= 3)
			{
				thrust::copy(block3.begin(), block3.end(), nGrams.begin());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());

				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());

			}

		}

	}

	else if (N == 5)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];

			std::cout << key;

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());

			st_block4 = lookUpitemMemory(iM, key);

			thrust::copy(st_block4.begin(), st_block4.end(), block4.begin());

			if (j >= 4)
			{
				thrust::copy(block4.begin(), block4.end(), nGrams.begin());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());


				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());

			}

		}

	}

	else if (N == 6)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());
			thrust::copy(block5.begin() + 1, block5.end(), block4.begin());

			st_block5 = lookUpitemMemory(iM, key);

			thrust::copy(st_block5.begin(), st_block5.end(), block5.begin());

			if (j >= 5)
			{
				thrust::copy(block5.begin(), block5.end(), nGrams.begin());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block4.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());

				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());

			}

		}

	}

	else if (N == 7)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());
			thrust::copy(block5.begin() + 1, block5.end(), block4.begin());
			thrust::copy(block6.begin() + 1, block6.end(), block5.begin());

			st_block6 = lookUpitemMemory(iM, key);

			thrust::copy(st_block6.begin(), st_block6.end(), block6.begin());

			if (j >= 6)
			{
				thrust::copy(block6.begin(), block6.end(), nGrams.begin());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block5.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block4.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());

				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());

			}

		}

	}

	else if (N == 8)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int i = j;
			char key = buffer[i];

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());
			thrust::copy(block5.begin() + 1, block5.end(), block4.begin());
			thrust::copy(block6.begin() + 1, block6.end(), block5.begin());
			thrust::copy(block7.begin() + 1, block7.end(), block6.begin());

			st_block7 = lookUpitemMemory(iM, key);

			thrust::copy(st_block7.begin(), st_block7.end(), block7.begin());

			if (j >= 7)
			{
				thrust::copy(block7.begin(), block7.end(), nGrams.begin());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block6.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block5.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block4.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());

				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());

				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());

			}

		}

	}
	thrust::copy(d_sumHV.begin(), d_sumHV.end(), sumHV.begin());

	return sumHV;

}

//Builds Associative Memory From the training Files
std::map<std::string, std::vector<int>> buildLanguage(std::map<char, std::vector<int>> iM)
{
	std::map< std::string, std::vector<int>> langAM;
	std::vector<int> langHV(D);

	int count = 0;
	std::string langLabels[64];


	DIR *pdir = NULL;

	pdir = opendir("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\Topic Recognition P2\\Topic Recognition P2\\Training Files P2"); // "." will refer to the current directory

	struct dirent *pent = NULL;


	if (pdir == NULL) 

	{

		std::cout << "\nERROR! pdir could not be initialised correctly";

		exit(3);

	} 



	while (pent = readdir(pdir))

	{

		if (pent == NULL)

		{

			std::cout << "\nERROR! pent could not be initialised correctly";

			exit(3);

		}
		if (strcmp(pent->d_name, ".") != 0 && strcmp(pent->d_name, "..") != 0)

		{
			std::cout << pent->d_name << std::endl;
			std::string name = pent->d_name;
			langLabels[count] = name.substr(0, 4);

			std::vector<std::string> list{ "C:", "Users", "Mohammed Aashyk", "Documents", "Visual Studio 2015", "Projects", "Topic Recognition P2", "Topic Recognition P2", "Training Files P2", name };

			std::string joined = boost::algorithm::join(list, "\\");

			std::ifstream t(joined);

					std::string str;


					t.seekg(0, std::ios::end);
					size_t size = t.tellg();
					std::string buffer(size, ' ');
					t.seekg(0);
					t.read(&buffer[0], size);

					std::cout << "Training File:" << langLabels[count] << std::endl;

					langHV = computeSumHV(iM, size, buffer);
					langAM[(langLabels[count])] = binarizeHV(langHV);

					count += 1;
				
				
			
		}
		
	}
	return langAM;

	
}

//Recognize the hypervectors by comparing it with Associative Memory
double test(std::map<char, std::vector<int>> iM, std::map<std::string, std::vector<int>> langAM)
{
	double total = 0.0;
	double correct = 0.0;
	double accuracy = 0;
	double maxAngle, angle = 0;
	std::string predictLang;
	std::vector<int> textHV;

	std::string langLabels[64];
	langLabels[0] = "acq0";
	langLabels[1] = "acq1";
	langLabels[2] = "acq2";
	langLabels[3] = "acq3";
	langLabels[4] = "acq4";
	langLabels[5] = "acq5";
	langLabels[6] = "acq6";
	langLabels[7] = "acq7";
	langLabels[8] = "cru0";
	langLabels[9] = "cru1";
	langLabels[10] = "cru2";
	langLabels[11] = "cru3";
	langLabels[12] = "cru4";
	langLabels[13] = "cru5";
	langLabels[14] = "cru6";
	langLabels[15] = "cru7";
	langLabels[16] = "gra0";
	langLabels[17] = "gra1";
	langLabels[18] = "gra2";
	langLabels[19] = "gra3";
	langLabels[20] = "gra4";
	langLabels[21] = "gra5";
	langLabels[22] = "gra6";
	langLabels[23] = "gra7";
	langLabels[24] = "int0";
	langLabels[25] = "int1";
	langLabels[26] = "int2";
	langLabels[27] = "int3";
	langLabels[28] = "int4";
	langLabels[29] = "int5";
	langLabels[30] = "int6";
	langLabels[31] = "int7";
	langLabels[32] = "mon0";
	langLabels[33] = "mon1";
	langLabels[34] = "mon2";
	langLabels[35] = "mon3";
	langLabels[36] = "mon4";
	langLabels[37] = "mon5";
	langLabels[38] = "mon6";
	langLabels[39] = "mon7";
	langLabels[40] = "ear0";
	langLabels[41] = "ear1";
	langLabels[42] = "ear2";
	langLabels[43] = "ear3";
	langLabels[44] = "ear4";
	langLabels[45] = "ear5";
	langLabels[46] = "ear6";
	langLabels[47] = "ear7";
	langLabels[48] = "shi0";
	langLabels[49] = "shi1";
	langLabels[50] = "shi2";
	langLabels[51] = "shi3";
	langLabels[52] = "shi4";
	langLabels[53] = "shi5";
	langLabels[54] = "shi6";
	langLabels[55] = "shi7";
	langLabels[56] = "tra0";
	langLabels[57] = "tra1";
	langLabels[58] = "tra2";
	langLabels[59] = "tra3";
	langLabels[60] = "tra4";
	langLabels[61] = "tra5";
	langLabels[62] = "tra6";
	langLabels[63] = "tra7";

	DIR *pdir = NULL; 

	    pdir = opendir ("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\Topic Recognition P2\\Topic Recognition P2\\testing_texts"); // "." will refer to the current directory

	    struct dirent *pent = NULL;

	    if (pdir == NULL) 

	    {

	        std::cout << "\nERROR! pdir could not be initialised correctly";

	        exit (3);

	    } 

	 

	    while (pent = readdir (pdir))

	    {

	        if (pent == NULL)

	        { 

	            std::cout << "\nERROR! pent could not be initialised correctly";

	            exit (3);

	        }
			if (strcmp(pent->d_name, ".") != 0 && strcmp(pent->d_name, "..") != 0)

			{
				std::cout << pent->d_name << std::endl;
				std::string name = pent->d_name;
				std::string actualLabel = name.substr(0, 3);
				std::vector<std::string> list{ "C:", "Users", "Mohammed Aashyk", "Documents", "Visual Studio 2015", "Projects", "Topic Recognition P2", "Topic Recognition P2", "testing_texts", name };

				std::string joined = boost::algorithm::join(list, "\\");

				std::ifstream t(joined);
				std::string str;

				t.seekg(0, std::ios::end);
				size_t size = t.tellg();
				std::string buffer(size, ' ');
				t.seekg(0);
				t.read(&buffer[0], size);

				std::cout << "Loading test file:" << pent->d_name << std::endl;
				textHV = computeSumHV(iM, size, buffer);
				textHV = binarizeHV(textHV);
				maxAngle = -1;
				for (int i = 0; i < 64; i++)
				{
					angle = cosine_similarity(langAM[langLabels[i]], textHV);
					if (angle > maxAngle)
					{
						maxAngle = angle;
						predictLang = langLabels[i].substr(0, 3);
					}
					

				}
				if (predictLang == actualLabel)
				{
					correct = correct + 1.0;
				}
				else
				{
					std::cout << predictLang << "  -->  " << actualLabel <<  std::endl;
				}
			}

			total = total + 1.0;
	    }

	    closedir (pdir);

		accuracy = correct / total * 100;

		return accuracy;
}



}

int main()
{
	
	std::vector<int> rand;
	std::map<char, std::vector<int>> iM;
	std::map<std::string, std::vector<int>> langAM;
	double correct;
	iM = createItemMemory(iM);										//creates Item Memory to initaiate the program
	langAM = buildLanguage(iM);										//Builds the associative memory from the train files
	correct = test(iM, langAM);										//Compares the test documents with the associative memory
	std::cout << correct << "%" << std::endl << "Run Success!";	
		//Displays Accuracy
}



