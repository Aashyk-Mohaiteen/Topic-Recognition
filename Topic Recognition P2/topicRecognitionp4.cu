/** @file topicRecognition.c 
 *
 * @brief Hyper-Dimension computing based topic detector 
 * 
 * @Author Mohammed Aashyk Mohaiteen Hebsur Rahman
 *
 * NOTE: boost library must be installed and included in 
 * the Additional Include Directories of the project 
 * Properties for this code.
 *
 */

#include <iostream>
#include <math.h>
#include <conio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <map>
#include <iterator>
#include <fstream>
#include <streambuf>
#include <string>
#include <dirent.h>
#include <boost/algorithm/string.hpp>
#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/functional.h>
#include <random>


//Algorithmic parameters for size of HD vectors and N-Grams
//
const int HD_VECTOR_SIZE = 10000;
const int N_GRAMS = 3;

/*!
 * @brief Generate Hyper Vectors with random values
 * @return The Randomized Hyper Vector
 */
std::vector<int> genRandomHV()
{
	std::vector<int> 	randomIndex(HD_VECTOR_SIZE);
	std::vector<int> 	randomHV(HD_VECTOR_SIZE);
	std::mt19937 		r{ std::random_device{}() };
	
	if ((HD_VECTOR_SIZE % 2) == 1)
	{
		std::cout << "Dimension is odd";
	}
	else
	{
		for (int i = 0; i < HD_VECTOR_SIZE; i++)
		{
			randomIndex[i] = i;
		}
		std::shuffle(randomIndex.begin(), randomIndex.end(), r);
		for (int i = 0; i < HD_VECTOR_SIZE / 2; i++)
		{
			randomHV[randomIndex[i]] = 1;
		}
		for (int i = HD_VECTOR_SIZE / 2;i < HD_VECTOR_SIZE;i++)
		{
			randomHV[randomIndex[i]] = -1;
		}
	}
	return randomHV;
} /* genRandomHV() */

/*!
 * @brief Create the Item Memory for each alphabet from which the corresponding
 * HV values will be derived
 * @param[in] iM Item Memory
 * @return The Item Memory	
 */
std::map<char, std::vector<int>> createItemMemory(std::map<char, std::vector<int>> iM)
{	
	char alphabet = 'a';
	int counter = 0;
	while ( counter < 26 )
	{
		iM[alphabet] = genRandomHV();
		alphabet = alphabet + 1;
		counter = counter + 1;
	}
	iM[char(32)] = genRandomHV();
	return iM;
} /* createItemMemory() */

/*!
 * @brief Retrieve the Character's corresponding Hyper vector
 * @param[in] iM Item Memory
 * @param[in] key Alphabet whose item memory needs to be retrieved
 * @return The Randomized HyperVector
 */
std::vector<int> lookUpitemMemory(std::map<char, std::vector<int>> iM, char key)
{
	std::vector<int> randomHV(HD_VECTOR_SIZE);
	
	randomHV = iM[key];
	return randomHV;
} /* lookUpitemMemory() */

/*!
 * @brief Finding the Jaccard similarity between Two Hyper vectors
 * @param[in] A Hyper vector of current character in the string
 * @param[in] B Hyper vector of next character in the string
 * @return Similarity
 */
double jaccard_similarity(std::vector<int> A, std::vector<int> B) {
	double 	m[4] = { 0,0,0,0 };
	int 	check;
	
	for (unsigned int i = 0; i < HD_VECTOR_SIZE; i++) {
		check = A[i] + B[i];
		if (check == 2) {
			m[3]++;
		}
		else if (check == 0) {
			if (B[i] == 1) {
				m[1]++;
			}
			else {
				m[2]++;
			}
		}
		else if (check == -2) {
			m[0]++;
		}
	}
	return (m[3] / (m[1] + m[2] + m[3]));
} /* jaccard_similarity() */

/*!
 * @brief Converting the vectors into values +1 and -1 before comparison(jaccard)
 * @param[in] langHV Hyper vector of character in the string
 * @return Binarized Hyper Vector
 */
std::vector<int> binarizeHV(std::vector<int> langHV)
{
	int threshold = 0;

	for (int i = 0; i < HD_VECTOR_SIZE; i++)
	{
		if (langHV[i] > threshold)
		{
			langHV[i] = 1;
		}
		else
		{
			langHV[i] = -1;
		}
	}
	return langHV;
} /* binarizeHV() */

/*!
 * @brief Generates the hyper vector for each input text file
 * @param[in] langHV Hyper vector of character in the string
 * @return The Hyper Vector generated 
 * NOTE: thrust algorithm is used which allocates the memory in the device (i.e, GPU)
 * and performs all operations in the GPU
 */
std::vector<int> computeSumHV(std::map<char, std::vector<int>> iM, size_t bufferSize, std::string  buffer)
{
	thrust::device_vector<int> st_block2(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> st_block3(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> st_block4(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> st_block5(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> st_block6(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> st_block7(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> block0(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> block1(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> block2(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> block3(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> block4(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> block5(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> block6(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> block7(HD_VECTOR_SIZE, 1);
	thrust::device_vector<int> nGrams(HD_VECTOR_SIZE, 1);
	thrust::device_vector <int> d_sumHV(HD_VECTOR_SIZE, 0);
	std::vector<int> sumHV(HD_VECTOR_SIZE, 0);
	
	if (N_GRAMS == 3)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int 	i = j;
			char 	key = buffer[i];
			#if 0 
			std::cout << key;
			#endif
			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			st_block2 = lookUpitemMemory(iM, key);
			thrust::copy(st_block2.begin(), st_block2.end(), block2.begin());
			if (j >= 2)
			{
				thrust::copy(block2.begin(), block2.end(), nGrams.begin());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());
				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
			}
		}
	}

	else if (N_GRAMS == 4)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int 	i = j;
			char 	key = buffer[i];

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			st_block3 = lookUpitemMemory(iM, key);
			thrust::copy(st_block3.begin(), st_block3.end(), block3.begin());
			if (j >= 3)
			{
				thrust::copy(block3.begin(), block3.end(), nGrams.begin());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());
				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
			}
		}
	}

	else if (N_GRAMS == 5)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int 	i = j;
			char 	key = buffer[i];
			#if 0
			std::cout << key;
			#endif
			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());
			st_block4 = lookUpitemMemory(iM, key);
			thrust::copy(st_block4.begin(), st_block4.end(), block4.begin());
			if (j >= 4)
			{
				thrust::copy(block4.begin(), block4.end(), nGrams.begin());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());
				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
			}
		}
	}

	else if (N_GRAMS == 6)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int 	i = j;
			char 	key = buffer[i];

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());
			thrust::copy(block5.begin() + 1, block5.end(), block4.begin());
			st_block5 = lookUpitemMemory(iM, key);
			thrust::copy(st_block5.begin(), st_block5.end(), block5.begin());
			if (j >= 5)
			{
				thrust::copy(block5.begin(), block5.end(), nGrams.begin());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block4.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());
				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
			}
		}
	}

	else if (N_GRAMS == 7)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int 	i = j;
			char 	key = buffer[i];

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());
			thrust::copy(block5.begin() + 1, block5.end(), block4.begin());
			thrust::copy(block6.begin() + 1, block6.end(), block5.begin());
			st_block6 = lookUpitemMemory(iM, key);
			thrust::copy(st_block6.begin(), st_block6.end(), block6.begin());
			if (j >= 6)
			{
				thrust::copy(block6.begin(), block6.end(), nGrams.begin());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block5.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block4.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());
				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
			}
		}
	}
	else if (N_GRAMS == 8)
	{
		for (size_t j = 0; j < bufferSize; j++)
		{
			int 	i = j;
			char 	key = buffer[i];

			thrust::copy(block1.begin() + 1, block1.end(), block0.begin());
			thrust::copy(block2.begin() + 1, block2.end(), block1.begin());
			thrust::copy(block3.begin() + 1, block3.end(), block2.begin());
			thrust::copy(block4.begin() + 1, block4.end(), block3.begin());
			thrust::copy(block5.begin() + 1, block5.end(), block4.begin());
			thrust::copy(block6.begin() + 1, block6.end(), block5.begin());
			thrust::copy(block7.begin() + 1, block7.end(), block6.begin());
			st_block7 = lookUpitemMemory(iM, key);
			thrust::copy(st_block7.begin(), st_block7.end(), block7.begin());
			if (j >= 7)
			{
				thrust::copy(block7.begin(), block7.end(), nGrams.begin());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block6.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block5.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block4.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block3.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block2.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block1.begin(), thrust::multiplies<int>());
				thrust::transform(nGrams.begin(), nGrams.end(), nGrams.begin(), block0.begin(), thrust::multiplies<int>());
				thrust::transform(d_sumHV.begin(), d_sumHV.end(), nGrams.begin(), d_sumHV.begin(), thrust::plus<int>());
			}

		}

	}
	thrust::copy(d_sumHV.begin(), d_sumHV.end(), sumHV.begin());
	return sumHV;
} /* computeSumHV() */

/*!
 * @brief Training the Associative Memory From the training Files
 * @param[in] iM item Memory
 * @return The generated Associative memory
 */
std::map<std::string, std::vector<int>> buildLanguage(std::map<char, std::vector<int>> iM)
{
	std::map<std::string, std::vector<int>> 	langAM;
	std::vector<int> 							langHV(HD_VECTOR_SIZE);
	int 										count = 0;
	std::string 								langLabels[64];

	DIR *pdir = NULL;
	pdir = opendir("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\Topic Recognition P2\\Topic Recognition P2\\Training Files P2"); // "." will refer to the current directory
	struct dirent *pent = NULL;
	if (pdir == NULL) 
	{
		std::cout << "\nERROR! pdir could not be initialised correctly";
		exit(3);
	} 
	while (pent = readdir(pdir))
	{
		if (pent == NULL)
		{
			std::cout << "\nERROR! pent could not be initialised correctly";
			exit(3);
		}
		if (strcmp(pent->d_name, ".") != 0 && strcmp(pent->d_name, "..") != 0)
		{
			std::cout << pent->d_name << std::endl;
			std::string name = pent->d_name;
			langLabels[count] = name.substr(0, 4);
			std::vector<std::string> list{ "C:", "Users", "Mohammed Aashyk", "Documents", "Visual Studio 2015", "Projects", "Topic Recognition P2", "Topic Recognition P2", "Training Files P2", name };
			std::string joined = boost::algorithm::join(list, "\\");
			std::ifstream t(joined);
			std::string str;

			t.seekg(0, std::ios::end);
			size_t size = t.tellg();
			std::string buffer(size, ' ');
			t.seekg(0);
			t.read(&buffer[0], size);
			std::cout << "Training File:" << langLabels[count] << std::endl;
			langHV = computeSumHV(iM, size, buffer);
			langAM[(langLabels[count])] = binarizeHV(langHV);
			count += 1;
		}
	}
	return langAM;
} /* buildLanguage() */

/*!
 * @brief Testing the associative memory by comparing it with testing files
 * @param[in] iM item Memory
 * @return The accuracy of the associative memory
 */
double test(std::map<char, std::vector<int>> iM, std::map<std::string, std::vector<int>> langAM)
{
	double 				total = 0.0;
	double 				correct = 0.0;
	double 				accuracy = 0;
	double 				maxAngle, angle = 0;
	std::string 		predictLang;
	std::vector<int> 	textHV;
	std::string 		langLabels[64];
	std::string			tmp;
	
	for ( int i = 0; i < 63; i++ ){
		if ( i < 8 )
		{
			tmp = "acq";
			tmp += std::to_string(i);
			langLabels[i] =  tmp;
		}
		else if ( i < 16 )
		{
			tmp = "cru";
			tmp += std::to_string(i);
			langLabels[i] = tmp; 
		} 
		else if ( i < 24 )
		{
			tmp = "gra";
			tmp += std::to_string(i);
			langLabels[i] = tmp;
		} 
		else if ( i < 32 )
		{
			tmp = "int";
			tmp += std::to_string(i);
			langLabels[i] = tmp;
		} 
		else if ( i < 40 )
		{
			tmp = "mon";
			tmp += std::to_string(i);
			langLabels[i] = tmp;
		} 
		else if ( i < 48 )
		{
			tmp = "ear";
			tmp += std::to_string(i);
			langLabels[i] = tmp;
		} 
		else if ( i < 56 )
		{
			tmp = "shi";
			tmp += std::to_string(i);
			langLabels[i] = tmp;
		} 
		else if ( i < 64 )
		{
			tmp = "tra";
			tmp += std::to_string(i);
			langLabels[i] = tmp;
		} 
		else
		{
			break;
		}
	}

	DIR *pdir = NULL; 
	pdir = opendir ("C:\\Users\\Mohammed Aashyk\\Documents\\Visual Studio 2015\\Projects\\Topic Recognition P2\\Topic Recognition P2\\testing_texts"); // "." will refer to the current directory
	struct dirent *pent = NULL;

	if (pdir == NULL) 
	{
	    std::cout << "\nERROR! pdir could not be initialised correctly";
	    exit (3);
	} 
	while (pent = readdir (pdir))
	{
	    if (pent == NULL)
	    { 
	        std::cout << "\nERROR! pent could not be initialised correctly";
	        exit (3);
	    }
		if (strcmp(pent->d_name, ".") != 0 && strcmp(pent->d_name, "..") != 0)
		{
			std::cout << pent->d_name << std::endl;
			std::string name = pent->d_name;
			std::string actualLabel = name.substr(0, 3);
			std::vector<std::string> list{ "C:", "Users", "Mohammed Aashyk", "Documents", "Visual Studio 2015", "Projects", "Topic Recognition P2", "Topic Recognition P2", "testing_texts", name };
			std::string joined = boost::algorithm::join(list, "\\");
			std::ifstream t(joined);
			std::string str;
				
			t.seekg(0, std::ios::end);
			size_t size = t.tellg();
			std::string buffer(size, ' ');
			t.seekg(0);
			t.read(&buffer[0], size);

			std::cout << "Loading test file:" << pent->d_name << std::endl;
			textHV = computeSumHV(iM, size, buffer);
			textHV = binarizeHV(textHV);
			maxAngle = -1;
			for (int i = 0; i < 64; i++)
			{
				angle = jaccard_similarity(langAM[langLabels[i]], textHV);
				if (angle > maxAngle)
				{
					maxAngle = angle;
					predictLang = langLabels[i].substr(0, 3);
				}
			}
			if (predictLang == actualLabel)
			{
				correct = correct + 1.0;
			}
			else
			{
				std::cout << predictLang << "  -->  " << actualLabel <<  std::endl;
			}
		}
		total = total + 1.0;
	}
	closedir (pdir);
	accuracy = correct / total * 100;
	return accuracy;
} /* test() */



int main()
{
	std::vector<int> rand;
	std::map<char, std::vector<int>> iM;
	std::map<std::string, std::vector<int>> langAM;
	double correct;
	iM = createItemMemory(iM);										//creates Item Memory to initaiate the program
	langAM = buildLanguage(iM);										//Builds the associative memory from the train files
	correct = test(iM, langAM);										//Compares the test documents with the associative memory
	std::cout << correct << "%" << std::endl << "Run Success!";	
		//Displays Accuracy
}

/*** end of file ***/